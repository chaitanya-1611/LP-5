#include<iostream>
#include <hip/hip_runtime.h>



__global__ void addVectors(int *a, int *b, int *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main(){
    int n = 1000000;
    int size = n*sizeof(int);

    // Host memory allocation
    int *h_a = new int[n];
    int *h_b = new int[n];
    int *h_c = new int[n];

    // Device memory pointers
    int *d_a, *d_b, *d_c;

    // Initialize host arrays
    for(int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i*2;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    addVectors<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print first few elements
    for(int i = 0; i < 10; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << "...\n";

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}



