#include<hip/hip_runtime.h>
#include<iostream>

__global__ void mult(int *A, int *B, int *c, int N){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    if(row<N && col<N){
        int sum=0;
        for(int i=0; i<N; i++){
            sum+=A[row*N+i]*B[i*N+col];
        }
        c[row*N+col]=sum;
    }
}   

int main(){
    int N = 512;
    int size = N*N*sizeof(int);
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            h_A[i*N+j]=i*N+j;
            h_B[i*N+j]=j*N+i;
        }
    }
    hipMemcpy(d_A, h_A,size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B,size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(N/dimBlock.x, N/dimBlock.y);
    mult<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    for(int i=0;i<10;i++){
        for(int j=0;j<10;j++){
            std::cout<<h_C[i*N+j]<<" ";
        }
        std::cout<<std::endl;
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    
    hipHostFree(h_B);
    hipHostFree(h_C);
    return 0;
}




